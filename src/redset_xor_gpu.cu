#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <string.h>
#include <errno.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#include "mpi.h"

#include "kvtree.h"
#include "kvtree_util.h"
#include "kvtree_mpi.h"

#include "redset_io.h"
#include "redset_util.h"
#include "redset.h"
#include "redset_internal.h"

static __global__ void xor_gpu(unsigned char* a, unsigned char* b, int n)
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    a[i] ^= b[i];
  }
}

/* apply XOR redundancy scheme to dataset files */
int redset_xor_encode_gpu(
  const redset_base* d,
  redset_lofi rsf,
  const char* my_chunk_file,
  int fd_xor,
  size_t chunk_size)
{
  int rc = REDSET_SUCCESS;

 /* get pointer to XOR state structure */
  redset_xor* state = (redset_xor*) d->state;

  /* allocate buffer to read a piece of my file */
  unsigned char** host_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* host_buf = host_bufs[0];

  unsigned char* send_buf;
  unsigned char* recv_buf;
  hipMalloc(&send_buf, redset_mpi_buf_size);
  hipMalloc(&recv_buf, redset_mpi_buf_size);

  MPI_Request request[2];
  MPI_Status  status[2];

  /* XOR Reduce_scatter */
  size_t nread = 0;
  while (nread < chunk_size) {
    size_t count = chunk_size - nread;
    if (count > redset_mpi_buf_size) {
      count = redset_mpi_buf_size;
    }

    int chunk_id;
    for(chunk_id = d->ranks-1; chunk_id >= 0; chunk_id--) {
      /* read the next set of bytes for this chunk from my file into send_buf */
      if (chunk_id > 0) {
        int chunk_id_rel = (d->rank + d->ranks + chunk_id) % d->ranks;
        if (chunk_id_rel > d->rank) {
          chunk_id_rel--;
        }
        unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;
        if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
        {
          rc = REDSET_FAILURE;
        }
      } else {
        memset(host_buf, 0, count);
      }

      /* copy file data from host to device */
      hipMemcpy(send_buf, host_buf, count, hipMemcpyHostToDevice);

      /* TODO: XORing with unsigned long would be faster here (if chunk size is multiple of this size) */
      /* merge the blocks via xor operation */
      if (chunk_id < d->ranks-1) {
        int nthreads = 1024;
        int nblocks = (count + nthreads - 1) / nthreads;
        xor_gpu<<<nblocks, nthreads>>>(send_buf, recv_buf, count);
        hipDeviceSynchronize();
      }

      if (chunk_id > 0) {
        /* not our chunk to write, forward it on and get the next */
        MPI_Irecv(recv_buf, count, MPI_BYTE, state->lhs_rank, 0, d->comm, &request[0]);
        MPI_Isend(send_buf, count, MPI_BYTE, state->rhs_rank, 0, d->comm, &request[1]);
        MPI_Waitall(2, request, status);
      } else {
        /* copy data from device to host for writing */
        hipMemcpy(host_buf, send_buf, count, hipMemcpyDeviceToHost);

        /* write send block to send chunk file */
        if (redset_write_attempt(my_chunk_file, fd_xor, host_buf, count) != count) {
          rc = REDSET_FAILURE;
        }
      }
    }

    nread += count;
  }

  hipFree(recv_buf);
  hipFree(send_buf);
  recv_buf = NULL;
  send_buf = NULL;

  /* free the buffers */
  redset_buffers_free(1, &host_bufs);

  return rc;
}

/* decode XOR redundancy scheme to rebuild missing files */
int redset_xor_decode_gpu(
  const redset_base* d,
  int root,
  redset_lofi rsf,
  const char* xor_file,
  int fd_xor,
  size_t chunk_size)
{
  int rc = REDSET_SUCCESS;
  MPI_Status status[2];

  /* get pointer to XOR state structure */
  redset_xor* state = (redset_xor*) d->state;

  /* allocate buffer to read a piece of my file */
  unsigned char** host_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* host_buf = host_bufs[0];

  unsigned char* send_buf;
  unsigned char* recv_buf;
  hipMalloc(&send_buf, redset_mpi_buf_size);
  hipMalloc(&recv_buf, redset_mpi_buf_size);

  /* Pipelined XOR Reduce to root */
  unsigned long offset = 0;
  int chunk_id;
  for (chunk_id = 0; chunk_id < d->ranks; chunk_id++) {
    size_t nread = 0;
    while (nread < chunk_size) {
      size_t count = chunk_size - nread;
      if (count > redset_mpi_buf_size) {
        count = redset_mpi_buf_size;
      }

      if (root != d->rank) {
        /* read the next set of bytes for this chunk from my file into host_buf */
        if (chunk_id != d->rank) {
          /* for this chunk, read data from the logical file */
          if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
          {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
          offset += count;
        } else {
          /* for this chunk, read data from the XOR file */
          if (redset_read_attempt(xor_file, fd_xor, host_buf, count) != count) {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        }

        /* copy file data from host to device */
        hipMemcpy(send_buf, host_buf, count, hipMemcpyHostToDevice);

        /* if not start of pipeline, receive data from left and xor with my own */
        if (root != state->lhs_rank) {
          MPI_Recv(recv_buf, count, MPI_BYTE, state->lhs_rank, 0, d->comm, &status[0]);

          int nthreads = 1024;
          int nblocks = (count + nthreads - 1) / nthreads;
          xor_gpu<<<nblocks, nthreads>>>(send_buf, recv_buf, count);
          hipDeviceSynchronize();
        }

        /* send data to right-side partner */
        MPI_Send(send_buf, count, MPI_BYTE, state->rhs_rank, 0, d->comm);
      } else {
        /* root of rebuild, just receive incoming chunks and write them out */
        MPI_Recv(recv_buf, count, MPI_BYTE, state->lhs_rank, 0, d->comm, &status[0]);

        /* copy data from device to host for writing */
        hipMemcpy(host_buf, recv_buf, count, hipMemcpyDeviceToHost);

        /* if this is not my xor chunk, write data to normal file, otherwise write to my xor chunk */
        if (chunk_id != d->rank) {
          /* for this chunk, write data to the logical file */
          if (redset_lofi_pwrite(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
          {
            /* write failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
          offset += count;
        } else {
          /* for this chunk, write data from the XOR file */
          if (redset_write_attempt(xor_file, fd_xor, host_buf, count) != count) {
            /* write failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        }
      }

      nread += count;
    }
  }

  hipFree(recv_buf);
  hipFree(send_buf);
  recv_buf = NULL;
  send_buf = NULL;

  /* free the buffers */
  redset_buffers_free(1, &host_bufs);

  return rc;
}

#ifdef __cplusplus
} /* extern C */
#endif
