#include "hip/hip_runtime.h"
#ifdef __cplusplus
extern "C" {
#endif

#include <stdio.h>
#include <string.h>
#include <errno.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#include "mpi.h"

#include "redset_io.h"
#include "redset_util.h"
#include "redset.h"
#include "redset_internal.h"
#include "redset_reedsolomon_common.h"

static __global__ void add_gpu(unsigned char* a, unsigned char* b, int n)
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    a[i] ^= b[i];
  }
}

static __global__ void multadd_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count && coeff != 0) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = rbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = gf_log[coeff] + gf_log[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] ^= (unsigned char) gf_exp[sumlogs];
    }
  }
}

static __global__ void premultadd_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */
  __shared__ unsigned char premult[256];

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < 256) {
    if (coeff != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      if (i != 0) {
        int sumlogs = gf_log[coeff] + gf_log[i];
        if (sumlogs >= gf_size - 1) {
          sumlogs -= (gf_size - 1);
        }
        premult[i] = (unsigned char) gf_exp[sumlogs];
      } else {
        premult[i] = (unsigned char) 0;
      }
    } else {
      premult[i] = (unsigned char) 0;
    }
  }
  __syncthreads();

  //size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count) {
    int data = (int) rbuf[i];
    dbuf[i] ^= premult[data];
  }
}

static __global__ void multadd2_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */
  __shared__ unsigned char logs[256];
  __shared__ unsigned char exps[256];
  //__shared__ unsigned char exps[512];

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < 256) {
    logs[i] = (unsigned char) gf_log[i];
    exps[i] = (unsigned char) gf_exp[i];
  }
  //else if (i < 512) {
  //  exps[i] = (unsigned char) gf_exp[i - 255];
  //}
  __syncthreads();

  //size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  //if (i < count && coeff != 0) {
  if (i < count) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = rbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = logs[coeff] + logs[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] ^= (unsigned char) exps[sumlogs];
    }
  }
}

static __global__ void scale_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count && coeff != 0) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = dbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = gf_log[coeff] + gf_log[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] = (unsigned char) gf_exp[sumlogs];
    }
  }
}

/* apply ReedSolomon redundancy scheme to dataset files */
int redset_reedsolomon_encode_gpu(
  const redset_base* d,
  redset_lofi rsf,
  const char* chunk_file,
  int fd_chunk,
  size_t chunk_size)
{
  int i;

  int rc = REDSET_SUCCESS;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* get offset into file immediately following the header */
  off_t header_size = lseek(fd_chunk, 0, SEEK_CUR);

  /* allocate buffer to read a piece of my file */
  unsigned char** host_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* host_buf = host_bufs[0];

  /* copy GF log and exp tables to GPU */
  unsigned int* gf_log;
  unsigned int* gf_exp;
  size_t table_size = state->gf_size * sizeof(unsigned int);
  hipMalloc(&gf_log, table_size);
  hipMalloc(&gf_exp, table_size);
  hipMemcpy(gf_log, state->gf_log, table_size, hipMemcpyHostToDevice);
  hipMemcpy(gf_exp, state->gf_exp, table_size, hipMemcpyHostToDevice);

  /* allocate send and receive buffers, and data buffer to accumulate result */
  unsigned char* data_bufs;
  unsigned char* recv_bufs;
  unsigned char* send_bufs;
  hipMalloc((void**)&data_bufs, redset_mpi_buf_size * state->encoding);
  hipMalloc((void**)&recv_bufs, redset_mpi_buf_size * state->encoding);
  hipMalloc((void**)&send_bufs, redset_mpi_buf_size * 1);
  unsigned char* sbuf = send_bufs;

  /* we'll issue a send/recv for each encoding block in each step */
  MPI_Request* request = (MPI_Request*) REDSET_MALLOC(state->encoding * 2 * sizeof(MPI_Request));
  MPI_Status*  status  = (MPI_Status*)  REDSET_MALLOC(state->encoding * 2 * sizeof(MPI_Status));

  /* process all data for this chunk */
  size_t nread = 0;
  while (nread < chunk_size) {
    /* limit the amount of data we read from the file at a time */
    size_t count = chunk_size - nread;
    if (count > redset_mpi_buf_size) {
      count = redset_mpi_buf_size;
    }

    /* initialize our reduction buffers */
    hipMemset(data_bufs, 0, redset_mpi_buf_size * state->encoding);

    /* In each step below, we read a chunk from our data files,
     * and send that data to the k ranks responsible for encoding
     * the checksums.  In each step, we'll receive a sliver of
     * data for each of the k blocks this process is responsible
     * for encoding */
    int chunk_step;
    for (chunk_step = d->ranks - 1; chunk_step >= state->encoding; chunk_step--) {
      /* get the chunk id for the current chunk */
      int chunk_id = (d->rank + chunk_step) % d->ranks;

      /* compute offset to read from within our file */
      int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, chunk_id);
      unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;

      /* read data from our file into send buffer */
      if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
      {
        /* read failed, make sure we fail this rebuild */
        rc = REDSET_FAILURE;
      }

      /* TODO: send straight from host buffer to avoid memcpy */
      /* copy file data from host to device */
      hipMemcpy(sbuf, host_buf, count, hipMemcpyHostToDevice);

      /* send data from our file to k ranks, and receive
       * incoming data from k ranks */
      int k = 0;
      for (i = 0; i < state->encoding; i++) {
        /* distance we're sending or receiving in this round */
        int dist = d->ranks - chunk_step + i;

        /* receive data from the right */
        int rhs_rank = (d->rank + dist + d->ranks) % d->ranks;
        unsigned char* rbuf = recv_bufs + i * redset_mpi_buf_size;
        MPI_Irecv(rbuf, count, MPI_BYTE, rhs_rank, 0, d->comm, &request[k]);
        k++;

        /* send our data to the left */
        int lhs_rank = (d->rank - dist + d->ranks) % d->ranks;
        MPI_Isend(sbuf, count, MPI_BYTE, lhs_rank, 0, d->comm, &request[k]);
        k++;
      }

      /* wait for communication to complete */
      MPI_Waitall(k, request, status);

      /* encode received data into our reduction buffers */
      for (i = 0; i < state->encoding; i++) {
        /* compute rank that sent to us */
        int dist = d->ranks - chunk_step + i;
        int received_rank = (d->rank + dist + d->ranks) % d->ranks;

        /* encode received data using its corresponding matrix
         * coefficient and accumulate to our reductino buffer */
        int row = i + d->ranks;
        unsigned int coeff = state->mat[row * d->ranks + received_rank];
        unsigned char* dbuf = data_bufs + i * redset_mpi_buf_size;
        unsigned char* rbuf = recv_bufs + i * redset_mpi_buf_size;
        int nthreads = 1024;
        int nblocks = (count + nthreads - 1) / nthreads;
        multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
        //if (coeff != 0) {
        //  multadd2_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
        //}
        //premultadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
      }

      hipDeviceSynchronize();
    }

    /* write final encoded data to our chunk file */
    for (i = 0; i < state->encoding; i++) {
      unsigned char* dbuf = data_bufs + i * redset_mpi_buf_size;
      hipMemcpy(host_buf, dbuf, count, hipMemcpyDeviceToHost);

      off_t offset = i * chunk_size + nread + header_size;
      if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
        rc = REDSET_FAILURE;
      }

      if (redset_write_attempt(chunk_file, fd_chunk, host_buf, count) != count) {
        rc = REDSET_FAILURE;
      }
    }

    nread += count;
  }

  redset_free(&request);
  redset_free(&status);

  hipFree(data_bufs);
  hipFree(recv_bufs);
  hipFree(send_bufs);
  data_bufs = NULL;
  recv_bufs = NULL;
  send_bufs = NULL;

  hipFree(gf_exp);
  hipFree(gf_log);
  gf_exp = NULL;
  gf_log = NULL;

  /* free buffers */
  redset_buffers_free(1, &host_bufs);

  return rc;
}

void redset_rs_reduce_decode_gpu(
  int ranks,
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  int chunk_id,
  int received_rank,
  int missing,
  int* rows,
  int count,
  unsigned char* recv_buf,
  unsigned char* data_bufs_dev)
{
  int i;

  /* determine encoding block this rank is responsible for in this chunk */
  int received_enc = redset_rs_get_encoding_id(ranks, state->encoding, received_rank, chunk_id);
  if (received_enc < ranks) {
    /* the data we received from this rank constitues actual data,
     * so we need to encode it by adding it to our sum */
    for (i = 0; i < missing; i++) {
      /* identify row for the data buffer in the encoding matrix,
       * then select the matrix element for the given rank,
       * finally mutiply recieved data by that coefficient and add
       * it to the data buffer */
      int row = rows[i] + ranks;
      unsigned int coeff = state->mat[row * ranks + received_rank];

      unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
      int nthreads = 1024;
      int nblocks = (count + nthreads - 1) / nthreads;
      multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, recv_buf);
    }
  } else {
    /* in this case, the rank is responsible for holding a
     * checksum block */
    for (i = 0; i < missing; i++) {
      /* get encoding row for the current data buffer */
      int row = rows[i] + ranks;
      if (row == received_enc) {
        /* in this case, we have the checksum, just add it in */
        unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
        int nthreads = 1024;
        int nblocks = (count + nthreads - 1) / nthreads;
        add_gpu<<<nblocks, nthreads>>>(dbuf, recv_buf, count);
      } else {
        /* otherwise, this rank would have contributed
         * 0-data for this chunk and for the selected encoding row */
      }
    }
  }

  hipDeviceSynchronize();

  return;
}

/* computed product of v1 * v2 using log and inverse log table lookups */
static unsigned int gf_mult_table_gpu(const redset_reedsolomon* state, unsigned int v1, unsigned int v2)
{
  /* 0 times anything is 0, we treat this as a special case since
   * there is no entry for 0 in the log table below, since there
   * is no value of x such that 2^x = 0 */
  if (v1 == 0 || v2 == 0) {
    return 0;
  }

  /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
  int sumlogs = state->gf_log[v1] + state->gf_log[v2];
  if (sumlogs >= state->gf_size - 1) {
    sumlogs -= (state->gf_size - 1);
  }
  int prod = state->gf_exp[sumlogs];

#if 0
  if (v1 >= state->gf_size ||
      v2 >= state->gf_size ||
      sumlogs >= state->gf_size - 1)
  {
    printf("ERRROR!!!!!\n");  fflush(stdout);
  }
#endif

  return prod;
}

/* scales a row r in a coefficient matrix in mat of size (rows x cols)
 * and an array of count values given in buf by a constant value val */
static void scale_row_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* mat,  /* coefficient matrix */
  int rows,           /* number of rows in mat */
  int cols,           /* number of cols in mat */
  unsigned int val,   /* constant to multiply elements by */
  int r,              /* row within mat to be scaled by val */
  int count,          /* number of elements in buf */
  unsigned char* buf) /* list of values to be scaled by val */
{
  /* scale values across given row */
  int col;
  for (col = 0; col < cols; col++) {
    mat[r * cols + col] = gf_mult_table_gpu(state, val, mat[r * cols + col]);
  }

  /* scale all values in buffer */
  int nthreads = 1024;
  int nblocks = (count + nthreads - 1) / nthreads;
  scale_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, buf, val);

  return;
}

/* multiply row a by the constant val, and add to row b in matrix,
 * and multiply elements in bufa and add to bufb element wise */
static void mult_add_row_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* mat,
  int rows,
  int cols,
  unsigned int val,
  int a,
  int b,
  int count,
  unsigned char* bufa,
  unsigned char* bufb)
{
  /* no need to do anything if we've zero'd out the row we're adding */
  if (val == 0) {
    return;
  }

  /* multiply row a by val and add to row b */
  int col;
  for (col = 0; col < cols; col++) {
    mat[b * cols + col] ^= (unsigned char) gf_mult_table_gpu(state, val, mat[a * cols + col]);
  }

  /* multiply values in bufa by val and add to bufb */
  int nthreads = 1024;
  int nblocks = (count + nthreads - 1) / nthreads;
  multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, bufb, val, bufa);

  return;
}

/* given matrix in mat of size (rows x cols) swap columns a and b */
static void swap_columns_gpu(unsigned int* mat, int rows, int cols, int a, int b)
{
  /* nothing to do if source and destination columns are the same */
  if (a == b) {
    return;
  }

  /* otherwise march down row and swap elements between column a and b */
  int row;
  for (row = 0; row < rows; row++) {
    unsigned int val = mat[row * cols + a];
    mat[row * cols + a] = mat[row * cols + b];
    mat[row * cols + b] = val;
  }
}

/* solve for x in Ax = b, where A (given in m) is a matrix of size (missing x missing)
 * using Gaussian elimination to convert A into an identity matrix,
 * here x and b are really matrices of size [missing, count] for count number of
 * individual [missing, 1] vectors */
static void redset_rs_gaussian_solve_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* m,      /* coefficient matrix to be reduced to an identity matrix */
  int missing,          /* number of rows and columns in m */
  int count,            /* length of buf arrays */
  unsigned char* bufs)  /* at list of count values for each of the missing unknowns */
{
  /* zero out lower portion of matrix */
  int row;
  for (row = 0; row < missing; row++) {
    /* search for first element in current row that is non-zero */
    int col;
    int nonzero = row;
    for (col = row; col < missing; col++) {
      unsigned int val = m[row * missing + col];
      if (val > 0) {
        nonzero = col;
        break;
      }
    }

    /* swap columns to ensure we have a nonzero in current starting position */
    swap_columns_gpu(m, missing, missing, row, nonzero);

    /* scale current row to start with a 1 */
    unsigned int val = m[row * missing + row];
    if (val != 0) {
      unsigned int imult = state->gf_imult[val];
      unsigned char* dbuf = bufs + row * redset_mpi_buf_size;
      scale_row_gpu(state, gf_log, gf_exp, m, missing, missing, imult, row, count, dbuf);
      hipDeviceSynchronize();
    }

    /* subtract current row from each row below to zero out any leading 1 */
    int r;
    for (r = row + 1; r < missing; r++) {
      /* multiply the target row by the leading term and subtract from the current row */
      unsigned int val = m[r * missing + row];
      unsigned char* abuf = bufs + row * redset_mpi_buf_size;
      unsigned char* bbuf = bufs + r   * redset_mpi_buf_size;
      mult_add_row_gpu(state, gf_log, gf_exp, m, missing, missing, val, row, r, count, abuf, bbuf);
    }
    hipDeviceSynchronize();
  }

  /* zero out upper portion of matrix */
  for (row = missing - 1; row > 0; row--) {
    /* for each row, compute factor needed to cancel out entry in current column
     * multiply target row and subtract from current row */
    int r;
    for (r = row - 1; r >= 0; r--) {
      /* multiply the target row by the leading term and subtract from the current row */
      unsigned int val = m[r * missing + row];
      unsigned char* abuf = bufs + row * redset_mpi_buf_size;
      unsigned char* bbuf = bufs + r   * redset_mpi_buf_size;
      mult_add_row_gpu(state, gf_log, gf_exp, m, missing, missing, val, row, r, count, abuf, bbuf);
    }
    hipDeviceSynchronize();
  }

  return;
}

/* given a filemap, a redundancy descriptor, a dataset id, and a failed rank in my xor set,
 * rebuild files and add them to the filemap */
int redset_reedsolomon_decode_gpu(
  const redset_base* d,
  int missing,
  int* rebuild_ranks,
  int need_rebuild,
  redset_lofi rsf,
  const char* chunk_file,
  int fd_chunk,
  size_t chunk_size)
{
  int i;

  int rc = REDSET_SUCCESS;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* get offset into file immediately following the header */
  off_t header_size = lseek(fd_chunk, 0, SEEK_CUR);

  /* allocate buffer to compute result of encoding,
   * we need one for each missing rank */
  unsigned char** data_bufs = (unsigned char**) redset_buffers_alloc(missing, redset_mpi_buf_size);

  /* allocate buffer to read a piece of my file */
  unsigned char** host_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* host_buf = host_bufs[0];

  /* allocate buffer to read a piece of the recevied chunk file,
   * we might get a message from each rank */
  unsigned char** recv_bufs = (unsigned char**) redset_buffers_alloc(d->ranks, redset_mpi_buf_size);
  unsigned char* rbuf = recv_bufs[0];

  unsigned int* gf_log;
  unsigned int* gf_exp;
  size_t table_size = state->gf_size * sizeof(unsigned int);
  hipMalloc(&gf_log, table_size);
  hipMalloc(&gf_exp, table_size);
  hipMemcpy(gf_log, state->gf_log, table_size, hipMemcpyHostToDevice);
  hipMemcpy(gf_exp, state->gf_exp, table_size, hipMemcpyHostToDevice);

  unsigned char* data_bufs_dev;
  unsigned char* recv_bufs_dev;
  hipMalloc((void**)&data_bufs_dev, redset_mpi_buf_size * missing);
  hipMalloc((void**)&recv_bufs_dev, redset_mpi_buf_size * d->ranks);

  unsigned char* send_buf_dev;
  hipMalloc(&send_buf_dev, redset_mpi_buf_size);

  /* switch send/recv to use device buffers */
  rbuf = recv_bufs_dev;
  unsigned char* sbuf = send_buf_dev;

  /* this array will map from missing rank number to missing data segment id,
   * which falls in the range [0, d->ranks + state->encoding),
   * we'll have one value for each missing rank */
  int* unknowns = (int*) REDSET_MALLOC(missing * sizeof(int));

  /* we'll have each process solve for the chunk matching its rank number */
  int decode_chunk_id = d->rank;
  for (i = 0; i < missing; i++) {
    int missing_rank = rebuild_ranks[i];
    unknowns[i] = redset_rs_get_encoding_id(d->ranks, state->encoding, missing_rank, decode_chunk_id);
  }

  /* given the ids of the unknown values,
   * pick among the available encoding rows for the quickest solve */
  unsigned int* m = NULL;
  int* rows = NULL;
  redset_rs_gaussian_solve_identify_rows(state, state->mat, d->ranks, state->encoding,
    missing, unknowns, &m, &rows
  );

  /* make a copy of the matrix coeficients */
  unsigned int* mcopy = (unsigned int*) REDSET_MALLOC(missing * missing * sizeof(unsigned int));
  
  /* during the reduce-scatter phase, each process has 1 outstanding send/recv at a time,
   * at the end, each process sends data to each failed rank and failed ranks receive a
   * message from all ranks, this allocation is more than needed */
  int max_outstanding = (d->ranks + state->encoding) * 2;
  MPI_Request* request = (MPI_Request*) REDSET_MALLOC(max_outstanding * sizeof(MPI_Request));
  MPI_Status*  status  = (MPI_Status*)  REDSET_MALLOC(max_outstanding * sizeof(MPI_Status));

  /* process all data for this chunk */
  size_t nread = 0;
  while (nread < chunk_size) {
    /* limit the amount of data we read from the file at a time */
    size_t count = chunk_size - nread;
    if (count > redset_mpi_buf_size) {
      count = redset_mpi_buf_size;
    }

    /* initialize buffers to accumulate reduction results */
    hipMemset(data_bufs_dev, 0, redset_mpi_buf_size * missing);

    int step_id;
    for (step_id = 0; step_id < d->ranks; step_id++) {
      int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;
      int rhs_rank = (d->rank + step_id + d->ranks) % d->ranks;

      /* get id of chunk we'll be sending in this step */
      int chunk_id = (d->rank + step_id) % d->ranks;

      /* get row number of encoding matrix we used for this chunk */
      int enc_id = redset_rs_get_encoding_id(d->ranks, state->encoding, d->rank, chunk_id);

      /* prepare our input buffers for the reduction */
      if (! need_rebuild) {
        /* we did not fail, so we can read data from our files,
         * determine whether we read from data files or chunk file */
        if (enc_id < d->ranks) {
          /* compute offset to read from within our file */
          int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, chunk_id);
          unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;

          /* read data from our file */
          if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
          {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        } else {
          /* for this chunk, read data from the chunk file */
          off_t offset = (enc_id - d->ranks) * chunk_size + nread + header_size;
          if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
            /* seek failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
          if (redset_read_attempt(chunk_file, fd_chunk, host_buf, count) != count) {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        }
      } else {
        /* if we're rebuilding, initialize our send buffer with 0,
         * so that our input does not contribute to the result */
        memset(host_buf, 0, count);
      }

      /* pipelined reduce-scatter across ranks */
      if (step_id > 0) {
        /* TODO: send straight from host buffer to avoid memcpy */
        /* copy file data from host to device */
        hipMemcpy(sbuf, host_buf, count, hipMemcpyHostToDevice);

        /* exchange data with neighboring ranks */
        MPI_Irecv(rbuf, count, MPI_BYTE, lhs_rank, 0, d->comm, &request[0]);
        MPI_Isend(sbuf, count, MPI_BYTE, rhs_rank, 0, d->comm, &request[1]);
        MPI_Waitall(2, request, status);
      } else {
        /* if we're rebuilding, initialize our send buffer with 0,
         * so that our input does not contribute to the result */
        /* copy file data from host to device */
        hipMemcpy(rbuf, host_buf, count, hipMemcpyHostToDevice);
      }

      /* merge received blocks via xor operation */
      redset_rs_reduce_decode_gpu(d->ranks, state, gf_log, gf_exp, decode_chunk_id, lhs_rank, missing, rows, count, rbuf, data_bufs_dev);
    }

    /* at this point, we need to invert our m matrix to solve for unknown values,
     * we invert a copy because we need to do this operation multiple times */
    memcpy(mcopy, m, missing * missing * sizeof(unsigned int));
    redset_rs_gaussian_solve_gpu(state, gf_log, gf_exp, mcopy, missing, count, data_bufs_dev);
    for (i = 0; i < missing; i++) {
      unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
      hipMemcpy(data_bufs[i], dbuf, redset_mpi_buf_size, hipMemcpyDeviceToHost);
    }

    /* TODO: for large groups, we may want to add some flow control here */

    /* send back our results to the failed ranks, just let it all fly */
    int k = 0;

    /* if we need to rebuild, post a receive from every other rank,
     * we stagger them based on our rank to support a natural ring */
    if (need_rebuild) {
      for (step_id = 0; step_id < d->ranks; step_id++) {
        int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;
        MPI_Irecv(recv_bufs[lhs_rank], count, MPI_BYTE, lhs_rank, 0, d->comm, &request[k]);
        k++;
      }
    }

    /* send the segments we rebuilt to each failed rank */
    for (i = 0; i < missing; i++) {
      int missing_rank = rebuild_ranks[i];
      MPI_Isend(data_bufs[i], count, MPI_BYTE, missing_rank, 0, d->comm, &request[k]);
      k++;
    }

    /* wait for all comms to finish */
    MPI_Waitall(k, request, status);

    /* if we need to rebuild, we now have data we can write to our files */
    if (need_rebuild) {
      for (step_id = 0; step_id < d->ranks; step_id++) {
        /* pick a rank to walk through our file */
        int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;

        /* at this point, we have the final result in our data buffers,
         * so we can write it out to the files */
        int received_chunk_id = lhs_rank;
        int enc_id = redset_rs_get_encoding_id(d->ranks, state->encoding, d->rank, received_chunk_id);
        if (enc_id < d->ranks) {
          /* for this chunk, write data to the logical file */
          int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, received_chunk_id);
          unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;
          if (redset_lofi_pwrite(&rsf, recv_bufs[lhs_rank], count, offset) != REDSET_SUCCESS)
          {
            /* write failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        } else {
          /* write send block to chunk file */
          off_t offset = (enc_id - d->ranks) * chunk_size + nread + header_size;
          if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
            rc = REDSET_FAILURE;
          }
          if (redset_write_attempt(chunk_file, fd_chunk, recv_bufs[lhs_rank], count) != count) {
            rc = REDSET_FAILURE;
          }
        }
      }
    }

    nread += count;
  }

  /* free off MPI requests */
  redset_free(&request);
  redset_free(&status);

  /* free matrix coefficients and selected rows needed to decode */
  redset_free(&mcopy);
  redset_free(&m);
  redset_free(&rows);

  hipFree(data_bufs_dev);
  hipFree(recv_bufs_dev);
  hipFree(send_buf_dev);
  hipFree(gf_exp);
  hipFree(gf_log);
  data_bufs_dev = NULL;
  recv_bufs_dev = NULL;
  send_buf_dev = NULL;
  gf_exp = NULL;
  gf_log = NULL;

  /* free buffers */
  redset_buffers_free(missing,  &data_bufs);
  redset_buffers_free(1,        &host_bufs);
  redset_buffers_free(d->ranks, &recv_bufs);

  return rc;
}

#ifdef __cplusplus
} /* extern C */
#endif
