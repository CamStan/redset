#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <errno.h>

#include <sys/types.h>
#include <sys/stat.h>
#include <fcntl.h>

#include "mpi.h"

#include "kvtree.h"
#include "kvtree_util.h"
#include "kvtree_mpi.h"

#include "redset_io.h"
#include "redset_util.h"
#include "redset.h"
#include "redset_internal.h"
#include "redset_reedsolomon_common.h"

#define REDSET_KEY_COPY_RS_DESC  "DESC"
#define REDSET_KEY_COPY_RS_CHUNK "CHUNK"
#define REDSET_KEY_COPY_RS_CKSUM "CKSUM"

#define ENABLE_CUDA 1

/*
=========================================
Distribute and file rebuild functions
=========================================
*/

/* set chunk filenames of form:  rs.<group_id>_<set_rank+1>_of_<set_ranks>.redset */
static void redset_build_rs_filename(
  const char* name,
  const redset_base* d,
  char* file, 
  size_t len)
{
  int rank_world;
  MPI_Comm_rank(d->parent_comm, &rank_world);
  snprintf(file, len, "%s%d.rs.grp_%d_of_%d.mem_%d_of_%d.redset",
    name, rank_world, d->group_id+1, d->groups, d->rank+1, d->ranks
  );
}

/* returns true if a an RS file is found for this rank,
 * sets xor_file to full filename */
static int redset_read_rs_file(
  const char* name,
  const redset_base* d,
  kvtree* header)
{
  /* set chunk filenames of form:  rs.<group_id>_<set_rank+1>_of_<set_ranks>.redset */
  char file[REDSET_MAX_FILENAME];
  redset_build_rs_filename(name, d, file, sizeof(file));

  /* check that we can read the file */
  if (redset_file_is_readable(file) != REDSET_SUCCESS) {
    redset_dbg(2, "Do not have read access to file: %s @ %s:%d",
      file, __FILE__, __LINE__
    );
    return REDSET_FAILURE;
  }

  /* read header info from file */
  if (kvtree_read_file(file, header) != KVTREE_SUCCESS) {
    return REDSET_FAILURE;
  }

  return REDSET_SUCCESS;
}

#define REDSET_KEY_COPY_RS_RANKS "RANKS"
#define REDSET_KEY_COPY_RS_GROUP "GROUP"
#define REDSET_KEY_COPY_RS_GROUP_RANK  "RANK"
#define REDSET_KEY_COPY_RS_GROUP_RANKS "RANKS"

/* given a redundancy descriptor with all top level fields filled in
 * allocate and fill in structure for Reed-Solomon specific fields in state */
int redset_construct_rs(MPI_Comm parent_comm, redset_base* d, int encoding)
{
  int rc = REDSET_SUCCESS;

  /* allocate a new structure to hold XOR state */
  redset_reedsolomon* state = (redset_reedsolomon*) REDSET_MALLOC(sizeof(redset_reedsolomon));

  /* attach structure to reddesc */
  d->state = (void*) state;

  /* allocate a new hash to store group mapping info */
  kvtree* header = kvtree_new();

  /* create a new empty hash to track group info for this xor set */
  kvtree* hash = kvtree_new();
  kvtree_set(header, REDSET_KEY_COPY_RS_GROUP, hash);

  /* record the total number of ranks in the set communicator */
  int ranks_comm;
  MPI_Comm_size(d->comm, &ranks_comm);
  kvtree_set_kv_int(hash, REDSET_KEY_COPY_RS_GROUP_RANKS, ranks_comm);

  /* record mapping of rank in set to corresponding parent rank */
  if (ranks_comm > 0) {
    /* allocate array to receive rank from each process */
    int* ranklist = (int*) REDSET_MALLOC(ranks_comm * sizeof(int));

    /* gather rank values */
    int parent_rank;
    MPI_Comm_rank(parent_comm, &parent_rank);
    MPI_Allgather(&parent_rank, 1, MPI_INT, ranklist, 1, MPI_INT, d->comm);

    /* map ranks in comm to ranks in comm */
    int i;
    for (i=0; i < ranks_comm; i++) {
      int rank = ranklist[i];
      kvtree_setf(hash, NULL, "%s %d %d", REDSET_KEY_COPY_RS_GROUP_RANK, i, rank);
    }

    /* free the temporary array */
    redset_free(&ranklist);
  }

  /* record group mapping info in descriptor */
  state->group_map = header;

  /* record group rank, world rank, and hostname of left and right partners */
  redset_set_partners(
    parent_comm, d->comm, 1,
    &state->lhs_rank, &state->lhs_rank_world, &state->lhs_hostname,
    &state->rhs_rank, &state->rhs_rank_world, &state->rhs_hostname
  );

  /* check that we got valid partners */
  if (state->lhs_hostname == NULL ||
      state->rhs_hostname == NULL ||
      strcmp(state->lhs_hostname, "") == 0 ||
      strcmp(state->rhs_hostname, "") == 0)
  {
    /* disable this descriptor */
    d->enabled = 0;
    redset_warn("Failed to find partner processes for redundancy descriptor, disabling @ %s:%d",
      __FILE__, __LINE__
    );
    rc = REDSET_FAILURE;
  } else {
    redset_dbg(2, "LHS partner: %s (%d)  -->  My name: %s (%d)  -->  RHS partner: %s (%d)",
      state->lhs_hostname, state->lhs_rank_world,
      redset_hostname, redset_rank,
      state->rhs_hostname, state->rhs_rank_world
    );
  }

  /* verify that all groups have a sufficient number of procs,
   * for the requested number of encoding blocks, number of
   * encoding blocks has to be less than number of procs in
   * each redundancy set */
  int valid = 1;
  if (encoding < 1 || encoding >= d->ranks) {
    valid = 0;
  }
  if (! redset_alltrue(valid, parent_comm)) {
    if (! valid) {
      redset_abort(-1, "Invalid number of Reed-Solomon encoding blocks (%d) for number of ranks in set %d @ %s:%d",
        encoding, d->ranks, __FILE__, __LINE__
      );
    }
  }

  /* allocate memory for Galois Field */
  int bits = 8;
  redset_rs_gf_alloc(state, d->ranks, encoding, bits);

  /* ensure that we're using a large enough Galois Field */
  valid = 1;
  if (d->ranks + encoding > state->gf_size) {
    /* we're going to need a bigger boat */
    valid = 0;
  }
  if (! redset_alltrue(valid, parent_comm)) {
    if (! valid) {
      redset_abort(-1, "More than %d bits required to encode %d ranks using %d encoding blocks @ %s:%d",
        bits, d->ranks, encoding, __FILE__, __LINE__
      );
    }
  }

  return rc;
}

int redset_delete_rs(redset_base* d)
{
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;
  if (state != NULL) {
    /* free the hash mapping group ranks to world ranks */
    kvtree_delete(&state->group_map);

    /* free strings that we received */
    redset_free(&state->lhs_hostname);
    redset_free(&state->rhs_hostname);

    /* free memory allocated for Galois Field structures */
    redset_rs_gf_delete(state);

    /* free the structure */
    redset_free(&d->state);
  }
  return REDSET_SUCCESS;
}

/* copy our redundancy descriptor info to a partner */
int redset_store_to_kvtree_rs(
  const redset_base* d,
  kvtree* hash)
{
  int rc = REDSET_SUCCESS;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* record number of encoding blocks */
  kvtree_util_set_int(hash, REDSET_KEY_COPY_RS_CKSUM, state->encoding);

  return rc;
}

/* this extracts parameters from the hash that are needed
 * in order to call create_rs */
int redset_read_from_kvtree_rs(
  const kvtree* hash,
  int* outencoding)
{
  int rc = REDSET_SUCCESS;

  /* record number of encoding blocks from hash */
  if (kvtree_util_get_int(hash, REDSET_KEY_COPY_RS_CKSUM, outencoding) != KVTREE_SUCCESS) {
    rc = REDSET_FAILURE;
  }

  return rc;
}

/* copy our redundancy descriptor info to a partner */
int redset_encode_reddesc_rs(
  kvtree* hash,
  const char* name,
  const redset_base* d)
{
  int rc = REDSET_SUCCESS;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* make a copy of the hash we want to encode */
  kvtree* send_hash = kvtree_new();
  kvtree_merge(send_hash, hash);

  /* we copy this hash to match the number of encoding blocks */
  int i;
  for (i = 1; i <= state->encoding; i++) {
    /* get ranks of procs to our left and right sides */
    int lhs_rank = (d->rank - i + d->ranks) % d->ranks;
    int rhs_rank = (d->rank + i + d->ranks) % d->ranks;

    /* send our redundancy descriptor hash to the right,
     * receive incoming hash from left neighbors */
    kvtree* partner_hash = kvtree_new();
    kvtree_sendrecv(send_hash, rhs_rank, partner_hash, lhs_rank, d->comm);
     
    /* store partner hash in our under its name */
    kvtree_merge(hash, partner_hash);
    kvtree_delete(&partner_hash);
  }

  /* delete our copy */
  kvtree_delete(&send_hash);

  return rc;
}

#if ENABLE_CUDA
__global__ void add_gpu(unsigned char* a, unsigned char* b, int n)
{
  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < n) {
    a[i] ^= b[i];
  }
}

__global__ void multadd_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count && coeff != 0) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = rbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = gf_log[coeff] + gf_log[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] ^= (unsigned char) gf_exp[sumlogs];
    }
  }
}

__global__ void premultadd_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */
  __shared__ unsigned char premult[256];

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < 256) {
    if (coeff != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      if (i != 0) {
        int sumlogs = gf_log[coeff] + gf_log[i];
        if (sumlogs >= gf_size - 1) {
          sumlogs -= (gf_size - 1);
        }
        premult[i] = (unsigned char) gf_exp[sumlogs];
      } else {
        premult[i] = (unsigned char) 0;
      }
    } else {
      premult[i] = (unsigned char) 0;
    }
  }
  __syncthreads();

  //size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count) {
    int data = (int) rbuf[i];
    dbuf[i] ^= premult[data];
  }
}

__global__ void multadd2_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff, unsigned char* rbuf)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */
  __shared__ unsigned char logs[256];
  __shared__ unsigned char exps[256];
  //__shared__ unsigned char exps[512];

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < 256) {
    logs[i] = (unsigned char) gf_log[i];
    exps[i] = (unsigned char) gf_exp[i];
  }
  //else if (i < 512) {
  //  exps[i] = (unsigned char) gf_exp[i - 255];
  //}
  __syncthreads();

  //size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  //if (i < count && coeff != 0) {
  if (i < count) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = rbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = logs[coeff] + logs[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] ^= (unsigned char) exps[sumlogs];
    }
  }
}

__global__ void scale_gpu(unsigned int* gf_log, unsigned int* gf_exp, int gf_size, size_t count, unsigned char* dbuf, unsigned int coeff)
{
  /* TODO: read gf_log into gf_exp thread-shared memory */

  size_t i = blockDim.x * blockIdx.x + threadIdx.x;
  if (i < count && coeff != 0) {
    /* 0 times anything is 0, we treat this as a special case since
     * there is no entry for 0 in the log table below, since there
     * is no value of x such that 2^x = 0 */
    int data = dbuf[i];
    if (data != 0) {
      /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
      int sumlogs = gf_log[coeff] + gf_log[data];
      if (sumlogs >= gf_size - 1) {
        sumlogs -= (gf_size - 1);
      }
      dbuf[i] = (unsigned char) gf_exp[sumlogs];
    }
  }
}
#endif

/* apply ReedSolomon redundancy scheme to dataset files */
int redset_apply_rs(
  int num_files,
  const char** files,
  const char* name,
  const redset_base* d)
{
  int rc = REDSET_SUCCESS;
  int i;

  /* pick out communicator */
  MPI_Comm comm = d->comm;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* compute number of data segments to divide user files by,
   * and check that we have enough ranks in our set */
  int num_segments = d->ranks - state->encoding;
  if (num_segments < 1) {
    redset_abort(-1, "Too few ranks in set (%d) for Reed-Solomon with %d encoding blocks @ %s:%d",
      d->ranks, state->encoding, __FILE__, __LINE__
    );
  }

  /* allocate buffers to hold reduction result, send buffer, and receive buffer */
  unsigned char** data_bufs = (unsigned char**) redset_buffers_alloc(state->encoding, redset_mpi_buf_size);
  unsigned char** recv_bufs = (unsigned char**) redset_buffers_alloc(state->encoding, redset_mpi_buf_size);

  /* allocate buffer to read a piece of my file */
  unsigned char** send_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* sbuf = send_bufs[0];

#if ENABLE_CUDA
  unsigned int* gf_log;
  unsigned int* gf_exp;
  size_t table_size = state->gf_size * sizeof(unsigned int);
  hipMalloc(&gf_log, table_size);
  hipMalloc(&gf_exp, table_size);
  hipMemcpy(gf_log, state->gf_log, table_size, hipMemcpyHostToDevice);
  hipMemcpy(gf_exp, state->gf_exp, table_size, hipMemcpyHostToDevice);

  unsigned char* data_bufs_dev;
  unsigned char* recv_bufs_dev;
  hipMalloc((void**)&data_bufs_dev, redset_mpi_buf_size * state->encoding);
  hipMalloc((void**)&recv_bufs_dev, redset_mpi_buf_size * state->encoding);

  unsigned char* send_buf_dev;
  hipMalloc(&send_buf_dev, redset_mpi_buf_size);

  /* switch send/recv to use device buffers */
  sbuf = send_buf_dev;
#endif

  /* use a host buffer for reading/writing to files */
  unsigned char* host_buf = send_bufs[0];

  /* allocate a structure to record meta data about our files and redundancy descriptor */
  kvtree* current_hash = kvtree_new();

  /* encode file info into hash */
  redset_lofi_encode_kvtree(current_hash, num_files, files);

  /* open logical file for reading */
  redset_lofi rsf;
  if (redset_lofi_open(current_hash, O_RDONLY, (mode_t)0, &rsf) != REDSET_SUCCESS) {
    redset_abort(-1, "Opening data files for reading for encoding @ %s:%d",
      __FILE__, __LINE__
    );
  }

  /* get size of our logical file */
  unsigned long my_bytes = redset_lofi_bytes(&rsf);

  /* store our redundancy descriptor in hash */
  kvtree* desc_hash = kvtree_new();
  redset_store_to_kvtree(d, desc_hash);
  kvtree_set(current_hash, REDSET_KEY_COPY_RS_DESC, desc_hash);

  /* create a hash to define our header information */
  kvtree* header = kvtree_new();

  /* record our rank within our redundancy group */
  kvtree_set_kv_int(header, REDSET_KEY_COPY_RS_GROUP_RANK, d->rank);

  /* copy meta data to hash */
  kvtree_setf(header, current_hash, "%s %d", REDSET_KEY_COPY_RS_DESC, d->rank);

  /* copy our descriptor N times to other ranks so it can be recovered
   * with to the same degree as our encoding scheme */
  for (i = 1; i <= state->encoding; i++) {
    /* get ranks of procs to our left and right sides */
    int lhs_rank = (d->rank - i + d->ranks) % d->ranks;
    int rhs_rank = (d->rank + i + d->ranks) % d->ranks;

    /* send our redundancy descriptor hash to the right,
     * receive incoming hash from left neighbors */
    kvtree* partner_hash = kvtree_new();
    kvtree_sendrecv(current_hash, rhs_rank, partner_hash, lhs_rank, comm);

    /* attach hash from this neighbor to our header */
    kvtree_setf(header, partner_hash, "%s %d", REDSET_KEY_COPY_RS_DESC, lhs_rank);
  }

  /* record the global ranks of the processes in our redundancy group */
  kvtree_merge(header, state->group_map);

  /* allreduce to get maximum filesize */
  unsigned long max_bytes;
  MPI_Allreduce(&my_bytes, &max_bytes, 1, MPI_UNSIGNED_LONG, MPI_MAX, comm);

  /* compute chunk size according to maximum file length and number segments,
   * if filesize doesn't divide evenly, then add one byte to chunk_size */
  size_t chunk_size = max_bytes / (unsigned long) num_segments;
  if ((unsigned long)num_segments * chunk_size < max_bytes) {
    chunk_size++;
  }

  /* TODO: need something like this to handle 0-byte files? */
  if (chunk_size == 0) {
    chunk_size++;
  }

  /* record the chunk size in the header */
  kvtree_util_set_bytecount(header, REDSET_KEY_COPY_RS_CHUNK, chunk_size);

  /* set chunk filenames of form:  rs.<group_id>_<set_rank+1>_of_<set_ranks>.redset */
  char chunk_file[REDSET_MAX_FILENAME];
  redset_build_rs_filename(name, d, chunk_file, sizeof(chunk_file));

  /* open my chunk file */
  mode_t mode_file = redset_getmode(1, 1, 0);
  int fd_chunk = redset_open(chunk_file, O_WRONLY | O_CREAT | O_TRUNC, mode_file);
  if (fd_chunk < 0) {
    /* TODO: try again? */
    redset_abort(-1, "Opening redundancy encoding file for writing: redset_open(%s) errno=%d %s @ %s:%d",
      chunk_file, errno, strerror(errno), __FILE__, __LINE__
    );
  }

  /* sort the header to list items alphabetically,
   * this isn't strictly required, but it ensures the kvtrees
   * are stored in the same byte order so that we can reproduce
   * the redundancy file identically on a rebuild */
  redset_sort_kvtree(header);

  /* write out the header */
  kvtree_write_fd(chunk_file, fd_chunk, header);
  kvtree_delete(&header);

  /* get offset into file immediately following the header */
  off_t header_size = lseek(fd_chunk, 0, SEEK_CUR);

  /* we'll issue a send/recv for each encoding block in each step */
  MPI_Request* request = (MPI_Request*) REDSET_MALLOC(state->encoding * 2 * sizeof(MPI_Request));
  MPI_Status*  status  = (MPI_Status*)  REDSET_MALLOC(state->encoding * 2 * sizeof(MPI_Status));

  /* process all data for this chunk */
  size_t nread = 0;
  while (nread < chunk_size) {
    /* limit the amount of data we read from the file at a time */
    size_t count = chunk_size - nread;
    if (count > redset_mpi_buf_size) {
      count = redset_mpi_buf_size;
    }

    /* initialize our reduction buffers */
#if ENABLE_CUDA
    hipMemset(data_bufs_dev, 0, redset_mpi_buf_size * state->encoding);
#else
    for (i = 0; i < state->encoding; i++) {
      memset(data_bufs[i], 0, count);
    }
#endif

    /* In each step below, we read a chunk from our data files,
     * and send that data to the k ranks responsible for encoding
     * the checksums.  In each step, we'll receive a sliver of
     * data for each of the k blocks this process is responsible
     * for encoding */
    int chunk_step;
    for (chunk_step = d->ranks - 1; chunk_step >= state->encoding; chunk_step--) {
      /* get the chunk id for the current chunk */
      int chunk_id = (d->rank + chunk_step) % d->ranks;

      /* compute offset to read from within our file */
      int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, chunk_id);
      unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;

      /* read data from our file into send buffer */
      if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
      {
        /* read failed, make sure we fail this rebuild */
        rc = REDSET_FAILURE;
      }

/* TODO: send straight from host buffer to avoid memcpy */
#if ENABLE_CUDA
      /* copy file data from host to device */
      hipMemcpy(sbuf, host_buf, count, hipMemcpyHostToDevice);
#else
      sbuf = host_buf;
#endif

      /* send data from our file to k ranks, and receive
       * incoming data from k ranks */
      int k = 0;
      for (i = 0; i < state->encoding; i++) {
        /* distance we're sending or receiving in this round */
        int dist = d->ranks - chunk_step + i;

        /* receive data from the right */
        int rhs_rank = (d->rank + dist + d->ranks) % d->ranks;
#if ENABLE_CUDA
        unsigned char* rbuf = recv_bufs_dev + i * redset_mpi_buf_size;
        MPI_Irecv(rbuf, count, MPI_BYTE, rhs_rank, 0, d->comm, &request[k]);
#else
        MPI_Irecv(recv_bufs[i], count, MPI_BYTE, rhs_rank, 0, d->comm, &request[k]);
#endif
        k++;

        /* send our data to the left */
        int lhs_rank = (d->rank - dist + d->ranks) % d->ranks;
        MPI_Isend(sbuf, count, MPI_BYTE, lhs_rank, 0, d->comm, &request[k]);
        k++;
      }

      /* wait for communication to complete */
      MPI_Waitall(k, request, status);

      /* encode received data into our reduction buffers */
      for (i = 0; i < state->encoding; i++) {
        /* compute rank that sent to us */
        int dist = d->ranks - chunk_step + i;
        int received_rank = (d->rank + dist + d->ranks) % d->ranks;

        /* encode received data using its corresponding matrix
         * coefficient and accumulate to our reductino buffer */
        int row = i + d->ranks;
        unsigned int coeff = state->mat[row * d->ranks + received_rank];
#if ENABLE_CUDA
        unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
        unsigned char* rbuf = recv_bufs_dev + i * redset_mpi_buf_size;
        int nthreads = 1024;
        int nblocks = (count + nthreads - 1) / nthreads;
        multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
        //if (coeff != 0) {
        //  multadd2_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
        //}
        //premultadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, rbuf);
#else
        redset_rs_reduce_buffer_multadd(state, count, data_bufs[i], coeff, recv_bufs[i]);
#endif
      }

#if ENABLE_CUDA
      hipDeviceSynchronize();
#endif
    }

    /* write final encoded data to our chunk file */
    for (i = 0; i < state->encoding; i++) {
#if ENABLE_CUDA
      unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
      hipMemcpy(data_bufs[i], dbuf, count, hipMemcpyDeviceToHost);
#endif

      off_t offset = i * chunk_size + nread + header_size;
      if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
        rc = REDSET_FAILURE;
      }

      if (redset_write_attempt(chunk_file, fd_chunk, data_bufs[i], count) != count) {
        rc = REDSET_FAILURE;
      }
    }

    nread += count;
  }

  /* close my chunkfile, with fsync */
  if (redset_close(chunk_file, fd_chunk) != REDSET_SUCCESS) {
    rc = REDSET_FAILURE;
  }

  /* close my dataset files */
  if (redset_lofi_close(&rsf) != REDSET_SUCCESS) {
    rc = REDSET_FAILURE;
  }

  redset_free(&request);
  redset_free(&status);

#if ENABLE_CUDA
  hipFree(data_bufs_dev);
  hipFree(recv_bufs_dev);
  hipFree(send_buf_dev);
  hipFree(gf_exp);
  hipFree(gf_log);
  data_bufs_dev = NULL;
  recv_bufs_dev = NULL;
  send_buf_dev = NULL;
  gf_exp = NULL;
  gf_log = NULL;
#endif

  /* free buffers */
  redset_buffers_free(state->encoding, &data_bufs);
  redset_buffers_free(state->encoding, &recv_bufs);
  redset_buffers_free(1,               &send_bufs);

#if 0
  /* if crc_on_copy is set, compute and store CRC32 value for chunk file */
  if (scr_crc_on_copy) {
    scr_compute_crc(map, id, scr_my_rank_world, my_chunk_file);
    /* TODO: would be nice to save this CRC in our partner's file so we can check correctness on a rebuild */
  }
#endif

  return rc;
}

#if ENABLE_CUDA
static void redset_rs_reduce_decode_gpu(
  int ranks,
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  int chunk_id,
  int received_rank,
  int missing,
  int* rows,
  int count,
  unsigned char* recv_buf,
  unsigned char* data_bufs_dev)
{
  int i;

  /* determine encoding block this rank is responsible for in this chunk */
  int received_enc = redset_rs_get_encoding_id(ranks, state->encoding, received_rank, chunk_id);
  if (received_enc < ranks) {
    /* the data we received from this rank constitues actual data,
     * so we need to encode it by adding it to our sum */
    for (i = 0; i < missing; i++) {
      /* identify row for the data buffer in the encoding matrix,
       * then select the matrix element for the given rank,
       * finally mutiply recieved data by that coefficient and add
       * it to the data buffer */
      int row = rows[i] + ranks;
      unsigned int coeff = state->mat[row * ranks + received_rank];

      unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
      int nthreads = 1024;
      int nblocks = (count + nthreads - 1) / nthreads;
      multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, dbuf, coeff, recv_buf);
    }
  } else {
    /* in this case, the rank is responsible for holding a
     * checksum block */
    for (i = 0; i < missing; i++) {
      /* get encoding row for the current data buffer */
      int row = rows[i] + ranks;
      if (row == received_enc) {
        /* in this case, we have the checksum, just add it in */
        unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
        int nthreads = 1024;
        int nblocks = (count + nthreads - 1) / nthreads;
        add_gpu<<<nblocks, nthreads>>>(dbuf, recv_buf, count);
      } else {
        /* otherwise, this rank would have contributed
         * 0-data for this chunk and for the selected encoding row */
      }
    }
  }

  hipDeviceSynchronize();

  return;
}

/* computed product of v1 * v2 using log and inverse log table lookups */
static unsigned int gf_mult_table_gpu(const redset_reedsolomon* state, unsigned int v1, unsigned int v2)
{
  /* 0 times anything is 0, we treat this as a special case since
   * there is no entry for 0 in the log table below, since there
   * is no value of x such that 2^x = 0 */
  if (v1 == 0 || v2 == 0) {
    return 0;
  }

  /* compute (v1 * v2) product as 2^( log_2(v1) + log_2(v2) ) in GF(2^bits) arithmetic */
  int sumlogs = state->gf_log[v1] + state->gf_log[v2];
  if (sumlogs >= state->gf_size - 1) {
    sumlogs -= (state->gf_size - 1);
  }
  int prod = state->gf_exp[sumlogs];

#if 0
  if (v1 >= state->gf_size ||
      v2 >= state->gf_size ||
      sumlogs >= state->gf_size - 1)
  {
    printf("ERRROR!!!!!\n");  fflush(stdout);
  }
#endif

  return prod;
}

/* scales a row r in a coefficient matrix in mat of size (rows x cols)
 * and an array of count values given in buf by a constant value val */
static void scale_row_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* mat,  /* coefficient matrix */
  int rows,           /* number of rows in mat */
  int cols,           /* number of cols in mat */
  unsigned int val,   /* constant to multiply elements by */
  int r,              /* row within mat to be scaled by val */
  int count,          /* number of elements in buf */
  unsigned char* buf) /* list of values to be scaled by val */
{
  /* scale values across given row */
  int col;
  for (col = 0; col < cols; col++) {
    mat[r * cols + col] = gf_mult_table_gpu(state, val, mat[r * cols + col]);
  }

  /* scale all values in buffer */
  int nthreads = 1024;
  int nblocks = (count + nthreads - 1) / nthreads;
  scale_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, buf, val);

  return;
}

/* multiply row a by the constant val, and add to row b in matrix,
 * and multiply elements in bufa and add to bufb element wise */
static void mult_add_row_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* mat,
  int rows,
  int cols,
  unsigned int val,
  int a,
  int b,
  int count,
  unsigned char* bufa,
  unsigned char* bufb)
{
  /* no need to do anything if we've zero'd out the row we're adding */
  if (val == 0) {
    return;
  }

  /* multiply row a by val and add to row b */
  int col;
  for (col = 0; col < cols; col++) {
    mat[b * cols + col] ^= (unsigned char) gf_mult_table_gpu(state, val, mat[a * cols + col]);
  }

  /* multiply values in bufa by val and add to bufb */
  int nthreads = 1024;
  int nblocks = (count + nthreads - 1) / nthreads;
  multadd_gpu<<<nblocks, nthreads>>>(gf_log, gf_exp, state->gf_size, count, bufb, val, bufa);

  return;
}

/* given matrix in mat of size (rows x cols) swap columns a and b */
static void swap_columns_gpu(unsigned int* mat, int rows, int cols, int a, int b)
{
  /* nothing to do if source and destination columns are the same */
  if (a == b) {
    return;
  }

  /* otherwise march down row and swap elements between column a and b */
  int row;
  for (row = 0; row < rows; row++) {
    unsigned int val = mat[row * cols + a];
    mat[row * cols + a] = mat[row * cols + b];
    mat[row * cols + b] = val;
  }
}

/* solve for x in Ax = b, where A (given in m) is a matrix of size (missing x missing)
 * using Gaussian elimination to convert A into an identity matrix,
 * here x and b are really matrices of size [missing, count] for count number of
 * individual [missing, 1] vectors */
static void redset_rs_gaussian_solve_gpu(
  redset_reedsolomon* state,
  unsigned int* gf_log,
  unsigned int* gf_exp,
  unsigned int* m,      /* coefficient matrix to be reduced to an identity matrix */
  int missing,          /* number of rows and columns in m */
  int count,            /* length of buf arrays */
  unsigned char* bufs)  /* at list of count values for each of the missing unknowns */
{
  /* zero out lower portion of matrix */
  int row;
  for (row = 0; row < missing; row++) {
    /* search for first element in current row that is non-zero */
    int col;
    int nonzero = row;
    for (col = row; col < missing; col++) {
      unsigned int val = m[row * missing + col];
      if (val > 0) {
        nonzero = col;
        break;
      }
    }

    /* swap columns to ensure we have a nonzero in current starting position */
    swap_columns_gpu(m, missing, missing, row, nonzero);

    /* scale current row to start with a 1 */
    unsigned int val = m[row * missing + row];
    if (val != 0) {
      unsigned int imult = state->gf_imult[val];
      unsigned char* dbuf = bufs + row * redset_mpi_buf_size;
      scale_row_gpu(state, gf_log, gf_exp, m, missing, missing, imult, row, count, dbuf);
      hipDeviceSynchronize();
    }

    /* subtract current row from each row below to zero out any leading 1 */
    int r;
    for (r = row + 1; r < missing; r++) {
      /* multiply the target row by the leading term and subtract from the current row */
      unsigned int val = m[r * missing + row];
      unsigned char* abuf = bufs + row * redset_mpi_buf_size;
      unsigned char* bbuf = bufs + r   * redset_mpi_buf_size;
      mult_add_row_gpu(state, gf_log, gf_exp, m, missing, missing, val, row, r, count, abuf, bbuf);
    }
    hipDeviceSynchronize();
  }

  /* zero out upper portion of matrix */
  for (row = missing - 1; row > 0; row--) {
    /* for each row, compute factor needed to cancel out entry in current column
     * multiply target row and subtract from current row */
    int r;
    for (r = row - 1; r >= 0; r--) {
      /* multiply the target row by the leading term and subtract from the current row */
      unsigned int val = m[r * missing + row];
      unsigned char* abuf = bufs + row * redset_mpi_buf_size;
      unsigned char* bbuf = bufs + r   * redset_mpi_buf_size;
      mult_add_row_gpu(state, gf_log, gf_exp, m, missing, missing, val, row, r, count, abuf, bbuf);
    }
    hipDeviceSynchronize();
  }

  return;
}
#endif

/* given a filemap, a redundancy descriptor, a dataset id, and a failed rank in my xor set,
 * rebuild files and add them to the filemap */
int redset_recover_rs_rebuild(
  const char* name,
  const redset_base* d,
  int missing,
  int* rebuild_ranks)
{
  int rc = REDSET_SUCCESS;
  int i;
  int j;

  redset_lofi rsf;
  int fd_chunk = -1;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* set chunk filename of form:  rs.<group_id>_<set_rank+1>_of_<set_ranks>.redset */
  char chunk_file[REDSET_MAX_FILENAME];
  redset_build_rs_filename(name, d, chunk_file, sizeof(chunk_file));

  /* allocate hash object to read in (or receive) the header of the redundancy file */
  kvtree* header = kvtree_new();

  /* TODO: pass this in as a parameter? */
  /* determine whether we need to rebuild */
  int need_rebuild = 0;
  for (i = 0; i < missing; i++) {
    if (rebuild_ranks[i] == d->rank) {
      /* we are one of the ranks who needs to rebuild our files */
      need_rebuild = 1;
    }
  }

  /* size of header as encoded in redundancy file */
  off_t header_size = 0;

  /* exchange headers and open each of our files for reading or writing */
  kvtree* current_hash = NULL;
  kvtree* send_hash = NULL;
  kvtree* recv_hash = NULL;
  if (! need_rebuild) {
    /* this process has all of its files,
     * open our redundancy file for reading */
    fd_chunk = redset_open(chunk_file, O_RDONLY);
    if (fd_chunk < 0) {
      redset_abort(-1, "Opening redundancy file for rebuild: redset_open(%s, O_RDONLY) errno=%d %s @ %s:%d",
        chunk_file, errno, strerror(errno), __FILE__, __LINE__
      );
    }

    /* read in the header */
    kvtree_read_fd(chunk_file, fd_chunk, header);

    /* get offset into file immediately following the header */
    header_size = lseek(fd_chunk, 0, SEEK_CUR);

    /* get file info for this rank */
    current_hash = kvtree_getf(header, "%s %d", REDSET_KEY_COPY_RS_DESC, d->rank);

    /* lookup number of files this process wrote */
    if (redset_lofi_open(current_hash, O_RDONLY, (mode_t)0, &rsf) != REDSET_SUCCESS) {
      redset_abort(-1, "Failed to open data files for reading during rebuild @ %s:%d",
        __FILE__, __LINE__
      );
    }

    /* if failed rank is to my left, i have its file info, send it the header */
    send_hash = kvtree_new();
    recv_hash = kvtree_new();
    for (i = 1; i <= state->encoding; i++) {
      int lhs_rank = (d->rank - i + d->ranks) % d->ranks;
      for (j = 0; j < missing; j++) {
        if (lhs_rank == rebuild_ranks[j]) {
          kvtree* payload = kvtree_new();
          kvtree_merge(payload, header);
          kvtree_setf(send_hash, payload, "%d", lhs_rank);
        }
      }
    }
    kvtree_exchange(send_hash, recv_hash, d->comm);
    kvtree_delete(&recv_hash);
    kvtree_delete(&send_hash);
  } else {
    /* this process failed, read our metadata from another process
     * we get our header from any rank that might have a copy */
    send_hash = kvtree_new();
    recv_hash = kvtree_new();
    kvtree_exchange(send_hash, recv_hash, d->comm);

    /* get our descriptor from first entry we find,
     * they are all the same */
    kvtree_elem* desc_elem = kvtree_elem_first(recv_hash);
    int source_rank = kvtree_elem_key_int(desc_elem);
    kvtree* desc_hash = kvtree_elem_hash(desc_elem);
    kvtree_merge(header, desc_hash);

    kvtree_delete(&recv_hash);
    kvtree_delete(&send_hash);

    /* get our current hash from header we received */
    current_hash = kvtree_getf(header, "%s %d", REDSET_KEY_COPY_RS_DESC, d->rank);

    /* replace the rank id with our own */
    kvtree_util_set_int(header, REDSET_KEY_COPY_RS_GROUP_RANK, d->rank);

    /* unset descriptors for ranks other than our partners */
    desc_hash = kvtree_get(header, REDSET_KEY_COPY_RS_DESC);
    for (i = 0; i < state->encoding; i++) {
      /* step through entries the source rank would have */
      int lhs_rank = (source_rank - i + d->ranks) % d->ranks;

      /* don't delete our own entry */
      if (lhs_rank == d->rank) {
        continue;
      }

      /* TODO: do this more cleanly */
      /* have to define the rank as a string */
      char rankstr[1024];
      snprintf(rankstr, sizeof(rankstr), "%d", lhs_rank);

      /* now we can delete this entry */
      kvtree_unset(desc_hash, rankstr);
    }

    /* get permissions for file */
    mode_t mode_file = redset_getmode(1, 1, 0);

    /* get the number of files that we need to rebuild */
    if (redset_lofi_open(current_hash, O_WRONLY | O_CREAT | O_TRUNC, mode_file, &rsf) != REDSET_SUCCESS) {
      redset_abort(-1, "Failed to open data files for writing during rebuild @ %s:%d",
        __FILE__, __LINE__
      );
    }

    /* open my redundancy file for writing */
    fd_chunk = redset_open(chunk_file, O_WRONLY | O_CREAT | O_TRUNC, mode_file);
    if (fd_chunk < 0) {
      /* TODO: try again? */
      redset_abort(-1, "Opening redundancy file for writing in rebuild: redset_open(%s) errno=%d %s @ %s:%d",
        chunk_file, errno, strerror(errno), __FILE__, __LINE__
      );
    }
  }

  /* if failed rank is to my right, send it my file info so it can write its header */
  send_hash = kvtree_new();
  recv_hash = kvtree_new();
  for (i = 1; i <= state->encoding; i++) {
    int rhs_rank = (d->rank + i + d->ranks) % d->ranks;
    for (j = 0; j < missing; j++) {
      if (rhs_rank == rebuild_ranks[j]) {
        kvtree* payload = kvtree_new();
        kvtree_merge(payload, current_hash);
        kvtree_setf(send_hash, payload, "%d", rhs_rank);
      }
    }
  }
  kvtree_exchange(send_hash, recv_hash, d->comm);

  if (need_rebuild) {
    /* receive copy of file info from left-side partners,
     * we'll store a copy of their headers for redudancy */
    kvtree_elem* desc_elem;
    for (desc_elem = kvtree_elem_first(recv_hash);
         desc_elem != NULL;
         desc_elem = kvtree_elem_next(desc_elem))
    {
      /* get source rank that sent this descriptor */
      char* rank_key = kvtree_elem_key(desc_elem);

      /* get the descriptor that was sent to us */
      kvtree* desc_hash = kvtree_elem_hash(desc_elem);

      /* make a copy of it */
      kvtree* partner_hash = kvtree_new();
      kvtree_merge(partner_hash, desc_hash);

      /* attach the copy to our header */
      kvtree_setf(header, partner_hash, "%s %s", REDSET_KEY_COPY_RS_DESC, rank_key);
    }

    /* sort the header to list items alphabetically,
     * this isn't strictly required, but it ensures the kvtrees
     * are stored in the same byte order so that we can reproduce
     * the redundancy file identically on a rebuild */
    redset_sort_kvtree(header);

    /* write chunk file header */
    kvtree_write_fd(chunk_file, fd_chunk, header);

    /* get offset into file immediately following the header */
    header_size = lseek(fd_chunk, 0, SEEK_CUR);
  }

  kvtree_delete(&recv_hash);
  kvtree_delete(&send_hash);

  /* read the chunk size used to compute the redundancy data */
  unsigned long chunk_size;
  if (kvtree_util_get_unsigned_long(header, REDSET_KEY_COPY_RS_CHUNK, &chunk_size) != REDSET_SUCCESS) {
    redset_abort(-1, "Failed to read chunk size from redundancy file header %s @ %s:%d",
      chunk_file, __FILE__, __LINE__
    );
  }

  /* allocate buffer to compute result of encoding,
   * we need one for each missing rank */
  unsigned char** data_bufs = (unsigned char**) redset_buffers_alloc(missing, redset_mpi_buf_size);

  /* allocate buffer to read a piece of my file */
  unsigned char** send_bufs = (unsigned char**) redset_buffers_alloc(1, redset_mpi_buf_size);
  unsigned char* sbuf = send_bufs[0];

  /* allocate buffer to read a piece of the recevied chunk file,
   * we might get a message from each rank */
  unsigned char** recv_bufs = (unsigned char**) redset_buffers_alloc(d->ranks, redset_mpi_buf_size);
  unsigned char* rbuf = recv_bufs[0];

#if ENABLE_CUDA
  unsigned int* gf_log;
  unsigned int* gf_exp;
  size_t table_size = state->gf_size * sizeof(unsigned int);
  hipMalloc(&gf_log, table_size);
  hipMalloc(&gf_exp, table_size);
  hipMemcpy(gf_log, state->gf_log, table_size, hipMemcpyHostToDevice);
  hipMemcpy(gf_exp, state->gf_exp, table_size, hipMemcpyHostToDevice);

  unsigned char* data_bufs_dev;
  unsigned char* recv_bufs_dev;
  hipMalloc((void**)&data_bufs_dev, redset_mpi_buf_size * missing);
  hipMalloc((void**)&recv_bufs_dev, redset_mpi_buf_size * d->ranks);

  unsigned char* send_buf_dev;
  hipMalloc(&send_buf_dev, redset_mpi_buf_size);

  /* switch send/recv to use device buffers */
  rbuf = recv_bufs_dev;
  sbuf = send_buf_dev;
#endif

  /* use a host buffer for reading/writing to files */
  unsigned char* host_buf = send_bufs[0];

  /* this array will map from missing rank number to missing data segment id,
   * which falls in the range [0, d->ranks + state->encoding),
   * we'll have one value for each missing rank */
  int* unknowns = (int*) REDSET_MALLOC(missing * sizeof(int));

  /* we'll have each process solve for the chunk matching its rank number */
  int decode_chunk_id = d->rank;
  for (i = 0; i < missing; i++) {
    int missing_rank = rebuild_ranks[i];
    unknowns[i] = redset_rs_get_encoding_id(d->ranks, state->encoding, missing_rank, decode_chunk_id);
  }

  /* given the ids of the unknown values,
   * pick among the available encoding rows for the quickest solve */
  unsigned int* m = NULL;
  int* rows = NULL;
  redset_rs_gaussian_solve_identify_rows(state, state->mat, d->ranks, state->encoding,
    missing, unknowns, &m, &rows
  );

  /* make a copy of the matrix coeficients */
  unsigned int* mcopy = (unsigned int*) REDSET_MALLOC(missing * missing * sizeof(unsigned int));
  
  /* during the reduce-scatter phase, each process has 1 outstanding send/recv at a time,
   * at the end, each process sends data to each failed rank and failed ranks receive a
   * message from all ranks, this allocation is more than needed */
  int max_outstanding = (d->ranks + state->encoding) * 2;
  MPI_Request* request = (MPI_Request*) REDSET_MALLOC(max_outstanding * sizeof(MPI_Request));
  MPI_Status*  status  = (MPI_Status*)  REDSET_MALLOC(max_outstanding * sizeof(MPI_Status));

  /* process all data for this chunk */
  size_t nread = 0;
  while (nread < chunk_size) {
    /* limit the amount of data we read from the file at a time */
    size_t count = chunk_size - nread;
    if (count > redset_mpi_buf_size) {
      count = redset_mpi_buf_size;
    }

    /* initialize buffers to accumulate reduction results */
#if ENABLE_CUDA
    hipMemset(data_bufs_dev, 0, redset_mpi_buf_size * missing);
#else
    for (i = 0; i < missing; i++) {
      memset(data_bufs[i], 0, count);
    }
#endif

    int step_id;
    for (step_id = 0; step_id < d->ranks; step_id++) {
      int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;
      int rhs_rank = (d->rank + step_id + d->ranks) % d->ranks;

      /* get id of chunk we'll be sending in this step */
      int chunk_id = (d->rank + step_id) % d->ranks;

      /* get row number of encoding matrix we used for this chunk */
      int enc_id = redset_rs_get_encoding_id(d->ranks, state->encoding, d->rank, chunk_id);

      /* prepare our input buffers for the reduction */
      if (! need_rebuild) {
        /* we did not fail, so we can read data from our files,
         * determine whether we read from data files or chunk file */
        if (enc_id < d->ranks) {
          /* compute offset to read from within our file */
          int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, chunk_id);
          unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;

          /* read data from our file */
          if (redset_lofi_pread(&rsf, host_buf, count, offset) != REDSET_SUCCESS)
          {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        } else {
          /* for this chunk, read data from the chunk file */
          off_t offset = (enc_id - d->ranks) * chunk_size + nread + header_size;
          if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
            /* seek failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
          if (redset_read_attempt(chunk_file, fd_chunk, host_buf, count) != count) {
            /* read failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        }
      } else {
        /* if we're rebuilding, initialize our send buffer with 0,
         * so that our input does not contribute to the result */
        memset(host_buf, 0, count);
      }

      /* pipelined reduce-scatter across ranks */
      if (step_id > 0) {
/* TODO: send straight from host buffer to avoid memcpy */
#if ENABLE_CUDA
        /* copy file data from host to device */
        hipMemcpy(sbuf, host_buf, count, hipMemcpyHostToDevice);
#else
        sbuf = host_buf;
#endif

        /* exchange data with neighboring ranks */
        MPI_Irecv(rbuf, count, MPI_BYTE, lhs_rank, 0, d->comm, &request[0]);
        MPI_Isend(sbuf, count, MPI_BYTE, rhs_rank, 0, d->comm, &request[1]);
        MPI_Waitall(2, request, status);
      } else {
        /* if we're rebuilding, initialize our send buffer with 0,
         * so that our input does not contribute to the result */
#if ENABLE_CUDA
        /* copy file data from host to device */
        hipMemcpy(rbuf, host_buf, count, hipMemcpyHostToDevice);
#else
        memcpy(rbuf, sbuf, count);
#endif
      }

      /* merge received blocks via xor operation */
#if ENABLE_CUDA
      redset_rs_reduce_decode_gpu(d->ranks, state, gf_log, gf_exp, decode_chunk_id, lhs_rank, missing, rows, count, rbuf, data_bufs_dev);
#else
      redset_rs_reduce_decode(d->ranks, state, decode_chunk_id, lhs_rank, missing, rows, count, rbuf, data_bufs);
#endif
    }

    /* at this point, we need to invert our m matrix to solve for unknown values,
     * we invert a copy because we need to do this operation multiple times */
    memcpy(mcopy, m, missing * missing * sizeof(unsigned int));
#if ENABLE_CUDA
    redset_rs_gaussian_solve_gpu(state, gf_log, gf_exp, mcopy, missing, count, data_bufs_dev);
    for (i = 0; i < missing; i++) {
      unsigned char* dbuf = data_bufs_dev + i * redset_mpi_buf_size;
      hipMemcpy(data_bufs[i], dbuf, redset_mpi_buf_size, hipMemcpyDeviceToHost);
    }
#else
    redset_rs_gaussian_solve(state, mcopy, missing, count, data_bufs);
#endif

    /* TODO: for large groups, we may want to add some flow control here */

    /* send back our results to the failed ranks, just let it all fly */
    int k = 0;

    /* if we need to rebuild, post a receive from every other rank,
     * we stagger them based on our rank to support a natural ring */
    if (need_rebuild) {
      for (step_id = 0; step_id < d->ranks; step_id++) {
        int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;
        MPI_Irecv(recv_bufs[lhs_rank], count, MPI_BYTE, lhs_rank, 0, d->comm, &request[k]);
        k++;
      }
    }

    /* send the segments we rebuilt to each failed rank */
    for (i = 0; i < missing; i++) {
      int missing_rank = rebuild_ranks[i];
      MPI_Isend(data_bufs[i], count, MPI_BYTE, missing_rank, 0, d->comm, &request[k]);
      k++;
    }

    /* wait for all comms to finish */
    MPI_Waitall(k, request, status);

    /* if we need to rebuild, we now have data we can write to our files */
    if (need_rebuild) {
      for (step_id = 0; step_id < d->ranks; step_id++) {
        /* pick a rank to walk through our file */
        int lhs_rank = (d->rank - step_id + d->ranks) % d->ranks;

        /* at this point, we have the final result in our data buffers,
         * so we can write it out to the files */
        int received_chunk_id = lhs_rank;
        int enc_id = redset_rs_get_encoding_id(d->ranks, state->encoding, d->rank, received_chunk_id);
        if (enc_id < d->ranks) {
          /* for this chunk, write data to the logical file */
          int chunk_id_rel = redset_rs_get_data_id(d->ranks, state->encoding, d->rank, received_chunk_id);
          unsigned long offset = chunk_size * (unsigned long) chunk_id_rel + nread;
          if (redset_lofi_pwrite(&rsf, recv_bufs[lhs_rank], count, offset) != REDSET_SUCCESS)
          {
            /* write failed, make sure we fail this rebuild */
            rc = REDSET_FAILURE;
          }
        } else {
          /* write send block to chunk file */
          off_t offset = (enc_id - d->ranks) * chunk_size + nread + header_size;
          if (redset_lseek(chunk_file, fd_chunk, offset, SEEK_SET) != REDSET_SUCCESS) {
            rc = REDSET_FAILURE;
          }
          if (redset_write_attempt(chunk_file, fd_chunk, recv_bufs[lhs_rank], count) != count) {
            rc = REDSET_FAILURE;
          }
        }
      }
    }

    nread += count;
  }

  /* free off MPI requests */
  redset_free(&request);
  redset_free(&status);

  /* free matrix coefficients and selected rows needed to decode */
  redset_free(&mcopy);
  redset_free(&m);
  redset_free(&rows);

  /* close my chunkfile */
  if (redset_close(chunk_file, fd_chunk) != REDSET_SUCCESS) {
    rc = REDSET_FAILURE;
  }

  /* close my checkpoint files */
  if (redset_lofi_close(&rsf) != REDSET_SUCCESS) {
    rc = REDSET_FAILURE;
  }

#if 0
  /* if i'm the rebuild rank, complete my file and chunk */
  if (root == d->rank) {
    /* complete each of our files and mark each as complete */
    for (i=0; i < num_files; i++) {
      /* TODO: need to check for errors, check that file is really valid */

      /* fill out meta info for our file and complete it */
      kvtree* meta_tmp = kvtree_get_kv_int(current_hash, REDSET_KEY_COPY_RS_FILE, i);

      /* TODODSET:write out filemap here? */

      /* if crc_on_copy is set, compute and store CRC32 value for each file */
      if (scr_crc_on_copy) {
        /* check for mismatches here, in case we failed to rebuild the file correctly */
        if (scr_compute_crc(map, id, scr_my_rank_world, filenames[i]) != REDSET_SUCCESS) {
          scr_err("Failed to verify CRC32 after rebuild on file %s @ %s:%d",
            filenames[i], __FILE__, __LINE__
          );

          /* make sure we fail this rebuild */
          rc = REDSET_FAILURE;
        }
      }
    }

    /* if crc_on_copy is set, compute and store CRC32 value for chunk file */
    if (scr_crc_on_copy) {
      /* TODO: would be nice to check for mismatches here, but we did not save this value in the partner file */
      scr_compute_crc(map, id, scr_my_rank_world, chunk_file);
    }
  }
#endif

  /* reapply metadata properties to file: uid, gid, mode bits, timestamps,
   * we do this on every file instead of just the rebuilt files so that we preserve atime on all files */
  redset_lofi_apply_meta(current_hash);

#if ENABLE_CUDA
  hipFree(data_bufs_dev);
  hipFree(recv_bufs_dev);
  hipFree(send_buf_dev);
  hipFree(gf_exp);
  hipFree(gf_log);
  data_bufs_dev = NULL;
  recv_bufs_dev = NULL;
  send_buf_dev = NULL;
  gf_exp = NULL;
  gf_log = NULL;
#endif

  /* free buffers */
  redset_buffers_free(missing,  &data_bufs);
  redset_buffers_free(1,        &send_bufs);
  redset_buffers_free(d->ranks, &recv_bufs);

  /* free the buffers */
  kvtree_delete(&header);

  return rc;
}

/* given a path, check whether files can be rebuilt via Reed-Solomon
 * and execute the rebuild if needed */
int redset_recover_rs(
  const char* name,
  const redset_base* d)
{
  MPI_Comm comm_world = d->parent_comm;

  /* get pointer to RS state structure */
  redset_reedsolomon* state = (redset_reedsolomon*) d->state;

  /* assume we have our files */
  int need_rebuild = 0;

  /* check whether we have our chunk file */
  kvtree* header = kvtree_new();
  if (redset_read_rs_file(name, d, header) == REDSET_SUCCESS) {
    /* got our chunk file, see if we have each data file */
    kvtree* current_hash = kvtree_getf(header, "%s %d", REDSET_KEY_COPY_RS_DESC, d->rank);
    if (redset_lofi_check(current_hash) != REDSET_SUCCESS) {
      /* some data file is bad */
      need_rebuild = 1;
    }
  } else {
    /* missing our chunk file */
    need_rebuild = 1;
  }
  kvtree_delete(&header);

  /* count how many in my set need to rebuild */
  int total_rebuild;
  MPI_Allreduce(&need_rebuild, &total_rebuild, 1, MPI_INT, MPI_SUM, d->comm);

  /* check whether all sets can rebuild, if not, bail out */
  int set_can_rebuild = (total_rebuild <= state->encoding);
  if (! redset_alltrue(set_can_rebuild, comm_world)) {
    return REDSET_FAILURE;
  }

  /* it's possible to rebuild; rebuild if we need to */
  int rc = REDSET_SUCCESS;
  if (total_rebuild > 0) {
    /* build list of members that need to rebuild */
    int* rebuild_ranks = (int*) REDSET_MALLOC(d->ranks * sizeof(int));

    /* someone in my set needs to rebuild, determine who */
    int tmp_rank = need_rebuild ? d->rank : -1;
    MPI_Allgather(&tmp_rank, 1, MPI_INT, rebuild_ranks, 1, MPI_INT, d->comm);

    /* slide ranks that need to be rebuilt to front of the array */
    int i;
    int slot = 0;
    for (i = 0; i < d->ranks; i++) {
      if (rebuild_ranks[i] != -1) {
        rebuild_ranks[slot] = i;
        slot++;
      }
    }

    /* rebuild */
    if (need_rebuild) {
      redset_dbg(2, "Rebuilding file from Reed-Solomon segments");
    }
    rc = redset_recover_rs_rebuild(name, d, total_rebuild, rebuild_ranks);

    /* free list of members that need to rebuild */
    redset_free(&rebuild_ranks);
  }

  /* check whether all sets rebuilt ok */
  if (! redset_alltrue(rc == REDSET_SUCCESS, comm_world)) {
    rc = REDSET_FAILURE;
  }

  return rc;
}

int redset_unapply_rs(
  const char* name,
  const redset_base* d)
{
  /* get name of reed-solomon file */
  char file[REDSET_MAX_FILENAME];
  redset_build_rs_filename(name, d, file, sizeof(file));

  int rc = redset_file_unlink(file);
  return rc;
}

/* returns a list of files added by redundancy descriptor */
redset_list* redset_filelist_get_rs(
  const char* name,
  redset_base* d)
{
  char file[REDSET_MAX_FILENAME];
  redset_build_rs_filename(name, d, file, sizeof(file));

  redset_list* list = (redset_list*) REDSET_MALLOC(sizeof(redset_list));
  list->count = 1;
  list->files = (const char**) REDSET_MALLOC(sizeof(char*));
  list->files[0] = strdup(file);

  return list;
}
